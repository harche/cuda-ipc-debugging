
#include <hip/hip_runtime.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <numeric>

#include <iostream>
#include <string>
#include <vector>

int main(int argc, char **argv)
{

    int n_gpus = 8;
    for (int i = 0; i < n_gpus; ++i) {
      for (int j = 0; j < n_gpus; ++j) {
        int result = -1;
        hipDeviceCanAccessPeer(&result, i, j);
        std::cerr<<"  - Can access peer "<<i<<", "<<j<<" : "<<result<<std::endl;
      }
    }

    return EXIT_SUCCESS;
}
