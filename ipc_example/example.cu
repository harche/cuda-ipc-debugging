// - process 1:
//  - ./example
//     -- if no shared memory handle is given
//        -  alloc gpu memory
//        -  create a shared memory handle
//        -  set gpu memory to 0
//        -  wait for other process to consume the data
//        -  ...
//        -  once data is consumed - exit
//
// - process 2:
// - ./example 0
//      - open shared memory handle 
//      - consume/modify data
//      - finish


// Parts taken from Nvidia's cuda-samples.


#include <hip/hip_runtime.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <numeric>
#include <cstdlib>

#include <iostream>
#include <string>
#include <vector>


#define DATA_SIZE 1024 * 1024 // 1MB

typedef struct shmStruct_st
{
    hipIpcMemHandle_t   memHandle;
    bool is_finished = false;
    bool handle_ready = false;
} shmStruct;

typedef struct sharedMemoryInfo_st {
    void *addr;
    size_t size;
    int shmFd;
} sharedMemoryInfo;


std::string const LSHM_NAME{"linux_shm"};

inline void __checkCUDAErrors(hipError_t err, const char *file, const int line) {
#if defined(DEBUG) || defined(_DEBUG)
    cudaDeviceSynchronize(); // Ensure errors from async calls are caught
#endif
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Runtime API error at %s:%d: %s.\n",
            file, line, hipGetErrorString(err));
        exit(EXIT_FAILURE); // Or throw an exception
    }
}

#define checkCudaErrors(err) __checkCUDAErrors (err, __FILE__, __LINE__)

int sharedMemoryCreate(const char *name, size_t sz, sharedMemoryInfo *info) {
  int status = 0;

  info->size = sz;

  info->shmFd = shm_open(name, O_RDWR | O_CREAT, 0777);
  if (info->shmFd < 0) {
    return errno;
  }

  status = ftruncate(info->shmFd, sz);
  if (status != 0) {
    return status;
  }

  info->addr = mmap(0, sz, PROT_READ | PROT_WRITE, MAP_SHARED, info->shmFd, 0);
  if (info->addr == NULL) {
    return errno;
  }

  return 0;
}

int sharedMemoryOpen(const char *name, size_t sz, sharedMemoryInfo *info) {
  info->size = sz;

  info->shmFd = shm_open(name, O_RDWR, 0777);
  if (info->shmFd < 0) {
    return errno;
  }

  info->addr = mmap(0, sz, PROT_READ | PROT_WRITE, MAP_SHARED, info->shmFd, 0);
  if (info->addr == NULL) {
    return errno;
  }

  return 0;
}

void sharedMemoryClose(sharedMemoryInfo *info) {
  if (info->addr) {
    munmap(info->addr, info->size);
  }
  if (info->shmFd) {
    close(info->shmFd);
  }
}

void producer() {

    shmStruct* shm = nullptr;

    // Make linux shared memory to share cudaIPCHandle
    sharedMemoryInfo info;
    if (sharedMemoryCreate(LSHM_NAME.c_str(), sizeof(shmStruct), &info) != 0) {
        printf("Failed to create shared memory slab\n");
        exit(EXIT_FAILURE);
    }

    shm = (shmStruct *)info.addr;
    memset((void *)shm, 0, sizeof(*shm));

    void* ptr = nullptr;
    checkCudaErrors(hipMalloc(&ptr, DATA_SIZE));
    checkCudaErrors(hipIpcGetMemHandle((hipIpcMemHandle_t *)&shm->memHandle, ptr));

    // set memory to zero in producer
    checkCudaErrors(hipMemset(ptr, 0, DATA_SIZE));
    
    // Signal that the handle is ready for consumer
    shm->handle_ready = true;
    std::cerr<<"IPC handle ready, entering wait loop ...\n";
    while (!shm->is_finished) {
    }

    std::vector<int8_t> cpu_data(DATA_SIZE, 0);
    checkCudaErrors(hipMemcpy(cpu_data.data(), ptr, DATA_SIZE, hipMemcpyDeviceToHost));

    auto const cpu_data_sum = std::accumulate(cpu_data.begin(), cpu_data.end(), 0);
    std::cerr<<"cpu_data sum "<<cpu_data_sum<<std::endl;

    sharedMemoryClose(&info);
}

void consumer(int const id) {
    // id isn't used anywhere. it is simply used in `main` to determine which
    // function to execute (i.e. producer / consumer)

    sharedMemoryInfo info;
    shmStruct* shm = nullptr;

    if (sharedMemoryOpen(LSHM_NAME.c_str(), sizeof(shmStruct), &info) != 0) {
        printf("Failed to open shared memory slab\n");
        exit(EXIT_FAILURE);
    }
    shm = (shmStruct *)info.addr;
    
    // Wait for producer to create and signal the IPC handle is ready
    std::cerr<<"Waiting for IPC handle to be ready...\n";
    while (!shm->handle_ready) {
        usleep(1000); // Sleep 1ms
    }
    std::cerr<<"IPC handle ready, opening memory handle...\n";
    
    void* ptr = nullptr;
    checkCudaErrors(
            hipIpcOpenMemHandle(&ptr, *(hipIpcMemHandle_t *)&shm->memHandle, hipIpcMemLazyEnablePeerAccess));

    hipMemset(ptr, 1, DATA_SIZE);

    hipDeviceSynchronize();
    shm->is_finished = true;
}

int main(int argc, char **argv)
{
    if (argc == 1) {
        producer();
    }
    else {
        consumer(atoi(argv[1]));
    }
    return EXIT_SUCCESS;
}
